#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "DeviceProperties.cuh"
#include "Header.cuh"

#define NUMBER float

using std::cout;
using namespace std;

const int height = 1024;
const int width = 2048;

__global__ void mandPixel(BYTE* image, int log2limit, NUMBER infRadius, NUMBER bottomLeftX, NUMBER bottomLeftY, NUMBER dx, NUMBER dy);

int main()
{
	system("title Mandelbrot set by Kozak Aron");
// GPU selection
	ListCUDADevices();
	hipSetDevice(0);
// allocation
    Measure Stopper = Measure();
    BYTE* image = new BYTE[3 * width * height];
    BYTE* gpu_image;
    hipMalloc((void**)&gpu_image, 3 * height * width * sizeof(BYTE));
    char* imageFileName = (char*)"bitmapImage.bmp";
    int BlockSize = 32;
    int GridSize = height * width / BlockSize; // 65536
// create mandelbrot:
    cout << "start" << endl;
    Stopper.start();
    int trialNum = 32;

    NUMBER bottomLeftX(-3), bottomLeftY(-1),  sizeX(4),  sizeY(2);
    const NUMBER infRadius = 4.0;
    int log2limit = 8;
    const NUMBER dx = sizeX / width;
    const NUMBER dy = sizeY / height;

    for (int trial = 0; trial < trialNum; trial++)
    {
        mandPixel << <GridSize, BlockSize >> > (gpu_image, log2limit, infRadius, bottomLeftX, bottomLeftY, dx, dy);
        hipDeviceSynchronize();
        hipMemcpy(image, gpu_image, 3 * height * width * sizeof(BYTE), hipMemcpyDeviceToHost);
    }

    long long int time = Stopper.stop(US);
    cout << "create mandelbrot: " << time / trialNum << " us" << endl;
// save mandelbrot:
    Stopper.start();
    generateBitmapImage(image, height, width, imageFileName);
    cout << "save bitmap image: " << Stopper.stop(MS) << " ms" << endl;
// free memory:
    delete[] image;
    hipFree(gpu_image);
    return 0;
}

__global__ void mandPixel(BYTE* image, int log2limit, NUMBER infRadius, NUMBER bottomLeftX, NUMBER bottomLeftY, NUMBER dx, NUMBER dy)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int c = threadID % width;
    int r = (threadID - c) / width;

    const unsigned int limit = (1 << log2limit) - 1;
    const NUMBER a_c = bottomLeftX + c * dx;
    const NUMBER b_c = bottomLeftY + r * dy;
    NUMBER a(0.0), b(0.0), temp;
    unsigned int iteration = 0;

    while ((++iteration < limit) && ((a * a + b * b) < infRadius))
    {
        temp = a * a - b * b + a_c;
        b = 2.0 * a * b + b_c;
        a = temp;
    }

    int pixel = 3 * threadID; //3 * (width * r + c);
    color(&image[pixel], iteration, log2limit);
}